#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"



__global__ void dynamic_para(int depth)
{
    printf("Depth : %d - tid : %d \n", depth, threadIdx.x);
    if(blockDim.x==1)
        return;
    
        if(threadIdx.x==0)
        {
            dynamic_para<<<1, blockDim.x/2>>>(depth+1);
        }
}

int main()
{
    dynamic_para<<<1, 16>>>(0);
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}