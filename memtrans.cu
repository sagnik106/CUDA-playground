#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void memtransf(int * arr)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("tid : %d, gid : %d, value : %d\n", threadIdx.x, gid, arr[gid]);
}

int main()
{
    int shape = 128;
    int size = shape * sizeof(int);

    int * h_arr;
    h_arr = (int *)malloc(size);

    for(int i=0; i<shape;i++)
    {
        h_arr[i]=(int)(rand() & 0xff);
    }

    int * d_arr;
    hipMalloc((void**)&d_arr, size);
    hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);

    dim3 block(64);
    dim3 grid(2);
    memtransf<<<grid, block>>>(d_arr);

    hipDeviceSynchronize();

    hipFree(d_arr);
    free(h_arr);

    hipDeviceReset();
    return 0;
}