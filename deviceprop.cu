#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;

int main()
{
    int n;
    hipGetDeviceCount(&n);
    cout<<"Number of CUDA enabled devices : "<<n<<endl;
    if(n!=0)
    {
        for(int i=0;i<n;i++)
        {
            cout<<"Device No. : "<<i<<endl;
            hipDeviceProp_t iProp;
            hipGetDeviceProperties(&iProp, i);
            cout<<"\tDevice Name               : "<<iProp.name<<endl;
            cout<<"\tNo. of multiprocessors    : "<<iProp.multiProcessorCount<<endl;
            cout<<"\tClock rate                : "<<iProp.clockRate<<" kHz"<<endl;
            cout<<"\tCoumpute Capability       : "<<iProp.major<<"."<<iProp.minor<<endl;
            cout<<"\tTotal Global Memory       : "<<iProp.totalGlobalMem<<" B"<<endl;
            cout<<"\tTotal Constant Memory     : "<<iProp.totalConstMem<<" B"<<endl;
            cout<<"\tShared Memory per Block   : "<<iProp.sharedMemPerBlock<<" B"<<endl;
            cout<<"\tRegisters per block       : "<<iProp.regsPerBlock<<endl;
            cout<<"\tWarp Size                 : "<<iProp.warpSize<<endl;
            cout<<"\tMaximum thread per block  : "<<iProp.maxThreadsPerBlock<<endl;
            cout<<"\tMaximum thread dimensions : ("<<iProp.maxThreadsDim[0]<<", "<<iProp.maxThreadsDim[1]<<", "<<iProp.maxThreadsDim[2]<<")"<<endl;
            cout<<"\tMaximum grid size         : ("<<iProp.maxGridSize[0]<<", "<<iProp.maxGridSize[1]<<", "<<iProp.maxGridSize[2]<<")"<<endl;
        }
    }
    return 0;
}