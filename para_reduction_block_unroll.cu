#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_unroll_block2(int * arr, int * temp, int l)
{
    int tid = threadIdx.x;
    int BLOCK_OFFSET = blockIdx.x * blockDim.x * 2;
    int index = BLOCK_OFFSET + tid;
    int * i_data = arr + BLOCK_OFFSET;
    if((index + blockDim.x) < l)
    {
        arr[index] += arr[index + blockDim.x];
    }
    __syncthreads();

    for(int offset=blockDim.x/2;offset>0;offset/=2)
    {
        if(tid<offset)
        {
            i_data[tid]+=i_data[tid+offset];
        }
        __syncthreads();
    }
    if(tid==0)
    {
        temp[blockIdx.x]=i_data[0];
    }
}

int cpu_summer(int * arr, int l)
{
    int s=0;
    for(int i=0;i<l;i++)
    {
        s+=arr[i];
    }
    return s;
}

int main()
{   
    int shape=1<<27;
    int size=shape*sizeof(int);
    int block_size=128;

    dim3 block(block_size);
    dim3 grid(shape/block.x/2);

    int * arr;
    arr=(int *)malloc(size);
    
    
    int temp_size=sizeof(int)*grid.x;
    int * tarr;
    tarr=(int *)malloc(temp_size);
    
    
    int * d_arr, * d_temp;
    hipMalloc((void**)&d_arr, size);
    hipMalloc((void**)&d_temp, temp_size);
    hipMemset(d_temp, 0, temp_size);


    for(int i=0; i< shape; i++)
    {
        arr[i]=(int)(rand() & 0x0f);
    }

    clock_t ct1,ct2,gt1,gt2;
    ct1=clock();
    int cpu=cpu_summer(arr, shape);
    ct2=clock();

    gt1=clock();
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);

    reduction_unroll_block2<<<grid, block>>>(d_arr, d_temp, shape);
    hipDeviceSynchronize();
    
    hipMemcpy(tarr, d_temp, temp_size, hipMemcpyDeviceToHost);

    int gpu=0;
    for(int i=0;i<grid.x;i++)
    {
        gpu+=tarr[i];
    }

    gt2=clock();

    printf(cpu==gpu?"CPU and GPU values Match\n":"CPU and GPU values do not match\n");
    printf("GPU time : %lf sec\n",(double)((gt2-gt1)/(double)CLOCKS_PER_SEC));
    printf("CPU time : %lf sec\n",(double)((ct2-ct1)/(double)CLOCKS_PER_SEC));

    hipFree(d_arr);
    hipFree(d_temp);
    free(arr);
    free(tarr);
    hipDeviceReset();

    return 0;
}