#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void adderboi(int * a, int * b, int * c)
{
    int gid=threadIdx.x + blockIdx.x * blockDim.x;
    c[gid] = a[gid] + b[gid];
}

int cpu_adder(int * a, int * b, int * c, int shape)
{
    for(int i=0;i<shape;i++)
    {
        if(c[i]!=a[i]+b[i])
            return 0;
    }
    return 1;
}

int main()
{
    int shape=1<<22;
    int size = shape*sizeof(int);
    int b;
    printf("Enter block size : ");
    scanf("%d",&b);
    dim3 grid(shape/b);

    int * h_arr1;
    int * h_arr2;
    int * h_arr3;

    h_arr1=(int *)malloc(size);
    h_arr2=(int *)malloc(size);
    h_arr3=(int *)malloc(size);

    for(int i=0; i< shape; i++)
    {
        h_arr1[i]=(int)(rand() & 0x0f);
        h_arr2[i]=(int)(rand() & 0x0f);
        h_arr3[i]=0;
    }

    int * d_arr1;
    int * d_arr2;
    int * d_arr3;

    hipMalloc((int**)&d_arr1, size);
    hipMalloc((int**)&d_arr2, size);
    hipMalloc((int**)&d_arr3, size);

    hipMemcpy(d_arr1, h_arr1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, h_arr2, size, hipMemcpyHostToDevice);

    adderboi <<<grid, b>>> (d_arr1, d_arr2, d_arr3);
    hipDeviceSynchronize();

    hipMemcpy(h_arr3, d_arr3, size, hipMemcpyDeviceToHost);

    printf(cpu_adder(h_arr1, h_arr2, h_arr3, shape)?"CPU and GPU values match\n":"CPU and GPU values donot match\n");
    /*for(int i=0;i<shape;i++)
    {
        printf("%d\t+\t%d\t= %d\n", h_arr1[i], h_arr2[i], h_arr3[i]);
    }*/

    hipFree(d_arr1);
    hipFree(d_arr2);
    hipFree(d_arr3);
    free(h_arr1);
    free(h_arr2);
    free(h_arr3);
    hipDeviceReset();
}