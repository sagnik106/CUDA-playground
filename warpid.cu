#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


__global__ void prt_details_wrp()
{
    int gid=blockIdx.y*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x;
    int warpid=threadIdx.x/32;
    int flatbid = blockIdx.y*gridDim.x+blockIdx.x;
    printf("gid : %d, warpid : %d, flattened bid : %d\n",gid, warpid,flatbid);
}

int main()
{
    dim3 block(42);
    dim3 grid(2, 2);
    prt_details_wrp <<<grid, block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}