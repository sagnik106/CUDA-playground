#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <cstring> //for memset

__global__ void adderboi(int * a, int * b, int * c)
{
    int gid=threadIdx.x + blockIdx.x * blockDim.x;
    c[gid] = a[gid] + b[gid];
}

int main()
{
    int shape=10;
    int size = shape*sizeof(int);

    int * h_arr1;
    int * h_arr2;
    int * h_arr3;

    h_arr1=(int *)malloc(size);
    h_arr2=(int *)malloc(size);
    h_arr3=(int *)malloc(size);

    for(int i=0; i< shape; i++)
    {
        h_arr1[i]=(int)(rand() & 0x0f);
        h_arr2[i]=(int)(rand() & 0x0f);
        h_arr3[i]=0;
    }

    int * d_arr1;
    int * d_arr2;
    int * d_arr3;

    hipMalloc((int**)&d_arr1, size);
    hipMalloc((int**)&d_arr2, size);
    hipMalloc((int**)&d_arr3, size);

    hipMemcpy(d_arr1, h_arr1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, h_arr2, size, hipMemcpyHostToDevice);

    adderboi <<<1, shape>>> (d_arr1, d_arr2, d_arr3);
    hipDeviceSynchronize();

    hipMemcpy(h_arr3, d_arr3, size, hipMemcpyDeviceToHost);

    for(int i=0;i<shape;i++)
    {
        printf("%d\t+\t%d\t= %d\n", h_arr1[i], h_arr2[i], h_arr3[i]);
    }

    hipFree(d_arr1);
    hipFree(d_arr2);
    hipFree(d_arr3);
    free(h_arr1);
    free(h_arr2);
    free(h_arr3);
    hipDeviceReset();
}