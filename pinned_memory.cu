#include <stdio.h>

#include "hip/hip_runtime.h"


void pined_memory()
{
    int shape = 1<<25;
    int size = shape * sizeof(float);

    float * h_a;
    hipHostMalloc((float **)&h_a, size, hipHostMallocDefault);

    float * d_a;
    hipMalloc((float **)&d_a, size);

    for(int i=0;i<shape;i++)
    {
        h_a[i]=6;
    }

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipHostFree(h_a);

    hipDeviceReset();
}

void paged_memory()
{
    int shape = 1<<25;
    int size = shape * sizeof(float);

    float * h_a = (float *)malloc(size);

    float * d_a;
    hipMalloc((float **)&d_a, size);

    for(int i=0;i<shape;i++)
    {
        h_a[i]=6;
    }

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    free(h_a);

    hipDeviceReset();
}

int main()
{
    int n;
    printf("Enter 1 for paged and 2 for pinned : ");
    scanf("%d",&n);
    switch(n)
    {
        case 1: paged_memory();break;
        case 2: pined_memory();break;
    }
    return 0;
}