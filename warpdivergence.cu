#include "hip/hip_runtime.h"


//#include "cuda_common.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void code_wo_divergence()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a=b=0;
    int warp_id = gid/32;
    if(warp_id %2==0)
    {
        a=100.0;
        b=50.0;
    }
    else
    {
        a=200.0;
        b=75.0;
    }
}

__global__ void code_w_divergence()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a=b=0;
    if(gid%2==0)
    {
        a=100.0;
        b=50.0;
    }
    else
    {
        a=200.0;
        b=75.0;
    }
}

int main()
{
    int size = 1<<22;
    dim3 block(128);
    dim3 grid((size+block.x-1)/block.x);

    code_wo_divergence <<<grid, block>>>();
    hipDeviceSynchronize();

    code_w_divergence <<<grid, block>>> ();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}