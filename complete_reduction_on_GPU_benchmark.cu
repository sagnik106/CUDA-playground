#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_neighbored_pairs(int * arr, int l, int offset)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if(gid>=offset)
        return;

    if(gid<offset)
    {
        arr[gid]+=arr[gid+offset];
        arr[gid+offset]=0;
    } 
}

int cpu_summer(int * arr, int l)
{
    int s=0;
    for(int i=0;i<l;i++)
    {
        s+=arr[i];
    }
    return s;
}

int main()
{   
    srand(time(0));
    int shape=1<<27;
    int size=shape*sizeof(int);
    int block_size=128;

    dim3 block(block_size);
    dim3 grid(shape>>1/block.x);

    int * arr;
    arr=(int *)malloc(size);
        
    int * d_arr;
    hipMalloc((void**)&d_arr, size);

    clock_t ct1,ct2,gt1,gt2,gtt1,gtt2;
    
    printf("CPU,GPU,GPU memory transfer");
    for(int counter=0;counter<200;counter++)
    {
        for(int i=0; i< shape; i++)
        {
            arr[i]=(int)(rand() & 0x0f);
        }

        ct1=clock();
        int cpu=cpu_summer(arr, shape);
        ct2=clock();

        gtt1=clock();
        hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
        gt1=clock();
        for(int offset=shape>>1;offset!=0;offset=offset>>1)
        {
            grid.x=offset>block.x?offset/block.x:1;
            reduction_neighbored_pairs<<<grid, block>>>(d_arr, shape, offset);
            hipDeviceSynchronize();
        }
        gt2=clock();
        hipMemcpy(arr, d_arr, sizeof(int), hipMemcpyDeviceToHost);
        gtt2=clock();

        printf("\n%lf,%lf,%lf",(double)((ct2-ct1)/(double)CLOCKS_PER_SEC),(double)((gt2-gt1)/(double)CLOCKS_PER_SEC),(double)((-gt2+gt1+gtt2-gtt1)/(double)CLOCKS_PER_SEC));
    }
    hipFree(d_arr);
    free(arr);
    hipDeviceReset();

    return 0;
}