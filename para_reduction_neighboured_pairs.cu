#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_neighbored_pairs(int * arr, int * temp, int l)
{
    int tid = threadIdx.x;
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(gid>l)
        return;

    for(int offset=1;offset<=blockDim.x/2;offset*=2)
    {
        if(tid%(2*offset)==0)
        {
            arr[gid]+=arr[gid+offset];
        }
        __syncthreads();
    }
    if(tid==0)
    {
        temp[blockIdx.x]=arr[gid];
    }
}

int cpu_summer(int * arr, int l)
{
    int s=0;
    for(int i=0;i<l;i++)
    {
        s+=arr[i];
    }
    return s;
}

int main()
{   
    int shape=1<<27;
    int size=shape*sizeof(int);
    int block_size=128;

    dim3 block(block_size);
    dim3 grid(shape/block.x);

    int * arr;
    arr=(int *)malloc(size);
    
    
    int temp_size=sizeof(int)*grid.x;
    int * tarr;
    tarr=(int *)malloc(temp_size);
    
    
    int * d_arr, * d_temp;
    hipMalloc((void**)&d_arr, size);
    hipMalloc((void**)&d_temp, temp_size);
    hipMemset(d_temp, 0, temp_size);


    for(int i=0; i< shape; i++)
    {
        arr[i]=(int)(rand() & 0x0f);
    }

    clock_t ct1,ct2,gt1,gt2;
    ct1=clock();
    int cpu=cpu_summer(arr, shape);
    ct2=clock();

    gt1=clock();
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);

    reduction_neighbored_pairs<<<grid, block>>>(d_arr, d_temp, shape);
    hipDeviceSynchronize();
    
    hipMemcpy(tarr, d_temp, temp_size, hipMemcpyDeviceToHost);

    int gpu=0;
    for(int i=0;i<grid.x;i++)
    {
        gpu+=tarr[i];
    }

    gt2=clock();

    printf(cpu==gpu?"CPU and GPU values Match\n":"CPU and GPU values do not match\n");
    printf("GPU time : %lf sec\n",(double)((gt2-gt1)/(double)CLOCKS_PER_SEC));
    printf("CPU time : %lf sec\n",(double)((ct2-ct1)/(double)CLOCKS_PER_SEC));

    hipFree(d_arr);
    hipFree(d_temp);
    free(arr);
    free(tarr);
    hipDeviceReset();

    return 0;
}