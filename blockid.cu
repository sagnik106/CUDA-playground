#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void print_threadIds()
{
    printf("blockIdx.x : %d blockIdx.y : %d blockIdx.z : %d gridDim.x : %d gridDim.y : %d gridDim.z : %d\n", blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z); 
}

int main()
{
    int nx=16, ny=16;

    dim3 block(8, 8);
    dim3 grid(nx/block.x, ny/block.y);
    print_threadIds <<<grid, block>>> ();
    hipDeviceSynchronize();
    hipDeviceReset();
}